#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>
#include <flashinfer/vec_dtypes.cuh>

using FP8_TYPE = c10::Float8_e4m3fn;
#define RMS_BLOCK_SIZE 256

// like std::array, but aligned
// goal: generate ld.128 and st.128 instructions
template <typename T, int sz>
struct __align__(alignof(T) * sz) array_t {
  T data[sz];
  using type = T;
  static constexpr int size = sz;
};

constexpr float max_8_bit = 448.0;
constexpr float min_8_bit = -448.0;

template <typename scalar_t>
__global__ void rms_norm_quant_kernel(scalar_t* __restrict__  input, scalar_t* __restrict__  weight,
        FP8_TYPE* __restrict__  output_q, float* __restrict__ output_s,
        const unsigned int d, const unsigned int rows, const float eps)
{
    int64_t row = blockIdx.x;
    int64_t tx = threadIdx.x;
    int64_t warp_id = tx/32;
    using P = array_t<scalar_t, 16 / sizeof(scalar_t)>;
    float acc = 0.f;
    __shared__ float reduction[RMS_BLOCK_SIZE/32];
    for(int64_t idx = tx; idx<d; idx+=blockDim.x)
    {
        P x = reinterpret_cast<P*>(input)[row * d + idx];

        for (int64_t i = 0; i<P::size; i++)
        {
            acc += (float)x.data[i] * (float)x.data[i];
        }

    }
    for (int mask = 16; mask>0; mask/=2)
    {
      acc += __shfl_xor_sync(0xffffffff, acc, mask, 32);
    }

    if(threadIdx.x%32 == 0)
    {
        reduction[warp_id] = acc;
    }

    __syncthreads();

    if (warp_id == 0)
    {
        acc = tx < RMS_BLOCK_SIZE/32 ? reduction[tx] : 0.f;
        acc += __shfl_xor_sync(0xffffffff, acc, 16, 32);
        acc += __shfl_xor_sync(0xffffffff, acc, 8, 32);
        acc += __shfl_xor_sync(0xffffffff, acc, 4, 32);
        acc += __shfl_xor_sync(0xffffffff, acc, 2, 32);
        acc += __shfl_xor_sync(0xffffffff, acc, 1, 32);
    }
    if(tx == 0)
    {
        float var = acc/(d*P::size);
        reduction[0] = rsqrtf(var + eps);
    }

    __syncthreads();
    acc = reduction[0];
    using O = array_t<FP8_TYPE, 8 / sizeof(FP8_TYPE)>;
    for(int64_t idx = tx; idx<d; idx+=blockDim.x)
    {
        float local_absmax = eps;
        P x = reinterpret_cast<P*>(input)[row * d + idx];
        P w = reinterpret_cast<P*>(weight)[idx];
        P interm;
        for (int64_t i = 0; i<P::size; i++)
        {
            interm.data[i] = (float)x.data[i] * acc;
            interm.data[i] *= (float)w.data[i];
            local_absmax = fmaxf(local_absmax, fabsf(interm.data[i]));
        }
        local_absmax = fmaxf(local_absmax, __shfl_xor_sync(0xffffffff, local_absmax, 8));
        local_absmax = fmaxf(local_absmax, __shfl_xor_sync(0xffffffff, local_absmax, 4));
        local_absmax = fmaxf(local_absmax, __shfl_xor_sync(0xffffffff, local_absmax, 2));
        local_absmax = fmaxf(local_absmax, __shfl_xor_sync(0xffffffff, local_absmax, 1));

        float y_s = (local_absmax/max_8_bit);
        if (threadIdx.x%16 == 0)
        {
             // = y_s;
            __stcg(&output_s[row*(d*P::size/128) + (idx * P::size) / 128], y_s);
        }

        O out;
        for (int64_t i = 0; i<P::size; i++)
        {
            float q = (float)interm.data[i]/y_s;
            float out_val = fminf(fmaxf(q, min_8_bit), max_8_bit);
            out.data[i] = FP8_TYPE(q);

        }
        __stcg(&reinterpret_cast<int2*>(output_q)[row * d + idx],
                *reinterpret_cast<int2*>(&out));
    }

}

template <typename scalar_t>
__global__ void rms_norm_quant_add_kernel(scalar_t* __restrict__  input,
        scalar_t* __restrict__  weight, scalar_t* __restrict__  residual,
        FP8_TYPE* __restrict__  output_q, float* __restrict__ output_s,
        const unsigned int d, const unsigned int rows, const float eps)
{
    int64_t row = blockIdx.x;
    int64_t tx = threadIdx.x;
    int64_t warp_id = tx/32;
    using P = array_t<scalar_t, 16 / sizeof(scalar_t)>;
    float acc = 0.f;
    __shared__ float reduction[RMS_BLOCK_SIZE/32];
    for(int64_t idx = tx; idx<d; idx+=blockDim.x)
    {
        P x = reinterpret_cast<P*>(input)[row * d + idx];
        P a = reinterpret_cast<P*>(residual)[row * d + idx];

        for (int64_t i = 0; i<P::size; i++)
        {
            x.data[i] += a.data[i];
            acc += (float)x.data[i] * (float)x.data[i];
        }
        reinterpret_cast<P*>(residual)[row * d + idx] = x;

    }
    for (int mask = 16; mask>0; mask/=2)
    {
      acc += __shfl_xor_sync(0xffffffff, acc, mask, 32);
    }

    if(threadIdx.x%32 == 0)
    {
        reduction[warp_id] = acc;
    }

    __syncthreads();

    if (warp_id == 0)
    {
        acc = tx < RMS_BLOCK_SIZE/32 ? reduction[tx] : 0.f;
        acc += __shfl_xor_sync(0xffffffff, acc, 16, 32);
        acc += __shfl_xor_sync(0xffffffff, acc, 8, 32);
        acc += __shfl_xor_sync(0xffffffff, acc, 4, 32);
        acc += __shfl_xor_sync(0xffffffff, acc, 2, 32);
        acc += __shfl_xor_sync(0xffffffff, acc, 1, 32);
    }
    if(tx == 0)
    {
        float var = acc/(d*P::size);
        reduction[0] = rsqrtf(var + eps);
    }

    __syncthreads();
    acc = reduction[0];
    using O = array_t<FP8_TYPE, 8 / sizeof(FP8_TYPE)>;
    for(int64_t idx = tx; idx<d; idx+=blockDim.x)
    {
        float local_absmax = eps;
        P x = reinterpret_cast<P*>(residual)[row * d + idx];
        P w = reinterpret_cast<P*>(weight)[idx];
        P interm;
        for (int64_t i = 0; i<P::size; i++)
        {
            interm.data[i] = (float)x.data[i] * acc;
            interm.data[i] *= (float)w.data[i];
            local_absmax = fmaxf(local_absmax, fabsf(interm.data[i]));
        }
        local_absmax = fmaxf(local_absmax, __shfl_xor_sync(0xffffffff, local_absmax, 8));
        local_absmax = fmaxf(local_absmax, __shfl_xor_sync(0xffffffff, local_absmax, 4));
        local_absmax = fmaxf(local_absmax, __shfl_xor_sync(0xffffffff, local_absmax, 2));
        local_absmax = fmaxf(local_absmax, __shfl_xor_sync(0xffffffff, local_absmax, 1));

        float y_s = (local_absmax/max_8_bit);
        if (threadIdx.x%16 == 0)
        {
            __stcg(&output_s[row*(d*P::size/128) + (idx * P::size) / 128], y_s);
        }

        O out;
        for (int64_t i = 0; i<P::size; i++)
        {
            float q = (float)interm.data[i]/y_s;
            float out_val = fminf(fmaxf(q, min_8_bit), max_8_bit);
            out.data[i] = FP8_TYPE(q);

        }
        __stcg(&reinterpret_cast<int2*>(output_q)[row * d + idx],
                *reinterpret_cast<int2*>(&out));
    }

}


void rms_norm_quant_launcher(torch::Tensor& input, torch::Tensor& output_q, torch::Tensor& output_s,
        torch::Tensor& weight, double eps)
{
    const unsigned int d = input.size(-1);
    const unsigned int rows = input.size(-2);

    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
                input.scalar_type(), "rms_norm_quant_add", ([&] {

                const unsigned int packed_d = std::ceil((float)d * sizeof(scalar_t) / 16);

                dim3 block_size = dim3(RMS_BLOCK_SIZE, 1, 1);
                dim3 grid_size = dim3(rows, 1, 1);

                const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
                const hipStream_t stream = at::cuda::getCurrentCUDAStream();

                rms_norm_quant_kernel<scalar_t><<<grid_size, block_size, 0, stream>>>
                (input.data_ptr<scalar_t>(), weight.data_ptr<scalar_t>(),
                 static_cast<FP8_TYPE*>(output_q.data_ptr()),
                 output_s.data_ptr<float>(),
                 packed_d, rows, eps);
                }));
}


void rms_norm_quant_add_launcher(torch::Tensor& input, torch::Tensor& residual,
        torch::Tensor& output_q, torch::Tensor& output_s,
        torch::Tensor& weight, double eps)
{
    const unsigned int d = input.size(-1);
    const unsigned int rows = input.size(-2);

    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
                input.scalar_type(), "rms_norm_quant_add", ([&] {

                const unsigned int packed_d = std::ceil((float)d * sizeof(scalar_t) / 16);

                dim3 block_size = dim3(RMS_BLOCK_SIZE, 1, 1);
                dim3 grid_size = dim3(rows, 1, 1);

                const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
                const hipStream_t stream = at::cuda::getCurrentCUDAStream();

                rms_norm_quant_add_kernel<scalar_t><<<grid_size, block_size, 0, stream>>>
                (input.data_ptr<scalar_t>(),
                 weight.data_ptr<scalar_t>(),
                 residual.data_ptr<scalar_t>(),
                 static_cast<FP8_TYPE*>(output_q.data_ptr()),
                 output_s.data_ptr<float>(),
                 packed_d, rows, eps);
                }));
}

