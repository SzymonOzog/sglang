#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>
#include <stdio.h>

void fused_moe_w8a8(
        const __hip_fp8_e4m3_fnuz* x,
        const float* x_scale,
        const __hip_fp8_e4m3_fnuz* w,
        const float* w_scale,
        __hip_bfloat16* out,
        const int* sorted_token_ids,
        const int* expert_ids,
        const int* num_tokens_post_padded,
        const int top_k,
        int M,
        int K,
        int N
        )
{
    printf("called fused moe yeah \n");
}
