#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>
#include <stdio.h>

// Not gonna type all that
using fp8 = __hip_fp8_e4m3_fnuz;

#define CP_ASYNC_CG(dst, src, Bytes) \
    asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(Bytes))

#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)

#define CP_ASYNC_WAIT_GROUP(N) asm volatile("cp.async.wait_group %0;\n" ::"n"(N))

__device__ __forceinline__ void ld_matrix_x2(uint32_t* tile, uint32_t mat)
{
    asm volatile("ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0, %1}, [%2];"
            : "=r"(tile[0]), "=r"(tile[1]) : "r"(mat));
}

__device__ __forceinline__ void ld_matrix_x4(uint32_t* tile, uint32_t mat)
{
    asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0, %1, %2, %3}, [%4];"
            : "=r"(tile[0]), "=r"(tile[1]), "=r"(tile[2]), "=r"(tile[3]) : "r"(mat));
}

#define S_BITS 3
#define S_MASK 0b1110000

template <int BM, int BK, int BN, int PF, int WM, int WN>
__global__ void fused_moe_w8a8_smem_kernel(
        const fp8* __restrict__ x,
        const float* __restrict__ x_scale,
        const fp8* __restrict__ w,
        const float* __restrict__ w_scale,
        __hip_bfloat16* __restrict__ out,
        const int* __restrict__ sorted_token_ids,
        const int* __restrict__ expert_ids,
        const int* __restrict__ num_tokens_post_padded,
        const int top_k,
        int M,
        int K,
        int N
        )
{
    const int32_t warpN = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpM = blockIdx.y*blockDim.y+threadIdx.y;

    //TODO should not be hardcoded
    constexpr int block_shape[2] = {128, 128};

    const int exp_idx = expert_ids[warpM];
    const fp8* exp_w = w + exp_idx * K * N;
    const int lane_id = threadIdx.x%32;
    const int warp_id = threadIdx.x/32;
    const int w_row = warpN * BN + (lane_id>>2);

    if(warpM * BM >= num_tokens_post_padded[0])
        return;

    // if(exp_idx < 0 || exp_idx >= 257)
    //     printf("INVALID IDX %d, %d, %d\n",blockIdx.y, exp_idx, num_tokens_post_padded[0]);


    int token_dest[2];
    token_dest[0] = sorted_token_ids[warpM*BM + (lane_id>>2)];
    token_dest[1] = sorted_token_ids[warpM*BM + (lane_id>>2) + 8];

    //SMEM sizes
    constexpr int WS = WN*PF*BK*BN;
    constexpr int XS = PF*BK*BM;
    // how many bytes we transfer per CP_ASYNC
    constexpr int TB = 16;
    // Thread offset per transfer
    constexpr int TO = TB/sizeof(fp8);
    __shared__ alignas(128) fp8 s_w[WS];
    __shared__ alignas(128) fp8 s_x[XS];

    uint32_t tile_x[4];
    uint32_t tile_w[2];
    float f_acc[4] = {0.f};
    int compute_stage=0;
    bool p = blockIdx.x == 1 && blockIdx.y == 128 && lane_id == 14;
    // bool p = blockIdx.x == 0 && blockIdx.y == 0 && lane_id == 0;
    auto load_tiles = [&](int off, int stage)
    {
            int xs_row = (lane_id>>2);
            if (token_dest[0]/top_k < M)
            {
                tile_x[0] = reinterpret_cast<const uint32_t*>(s_x + xs_row*PF*BK + stage*BK)[lane_id%4];
                tile_x[2] = reinterpret_cast<const uint32_t*>(s_x + xs_row*PF*BK + stage*BK + 16)[lane_id%4];
            }
            if (token_dest[1]/top_k < M)
            {
                xs_row += 8;
                tile_x[1] = reinterpret_cast<const uint32_t*>(s_x + xs_row*PF*BK + stage*BK)[lane_id%4];
                tile_x[3] = reinterpret_cast<const uint32_t*>(s_x + xs_row*PF*BK + stage*BK + 16)[lane_id%4];
            }
            // const int xs_col = (lane_id/16)*(BK/2) + stage*BK;
            // uint32_t sm_x = __cvta_generic_to_shared(s_x + xs_row*PF*BK + xs_col);
            // ld_matrix_x4(tile_x, sm_x);
            // if(p)
            //     printf("reading %d tile %d/%d, stage %d\n", token_dest[0], xs_row, xs_col, stage);

            const int ws_row = warp_id*BN + (lane_id%8);
            const int ws_col = (lane_id/8)*(BK/2) + stage*BK;
            int i = ws_row*PF*BK + ws_col;

            int swizzled = i^((i&(S_MASK<<S_BITS))>>S_BITS);
            // int swizzled=i;
            uint32_t sm_w = __cvta_generic_to_shared(s_w + swizzled);
            ld_matrix_x2(tile_w, sm_w);
            // if(p)
            //     printf("reading %d tile %d/%d, stage %d\n", token_dest[0], ws_row, ws_col, stage);
    };

    for (int block=0; block < K/block_shape[0]; block += 1)
    {
        const int scale_cols_x = K/block_shape[1];
        const int scale_rows_w = N/block_shape[1];
        const int scale_cols_w = K/block_shape[0];
        int b_off = block * block_shape[0];

        for(int i = (threadIdx.y*blockDim.x + threadIdx.x)*TO;
                i < WS;
                i += blockDim.x*blockDim.y*TO)
        {
            int row = blockIdx.x*WN*BN + i/(BK*PF);
            int col = b_off + i%(BK*PF);
            int swizzled = i^((i&(S_MASK<<S_BITS))>>S_BITS);
            // int swizzled=i;
            uint32_t sm = __cvta_generic_to_shared(s_w + swizzled);
            // if(p)
            //     printf("loading %d to tile %d %d, i %d\n", exp_idx, row, col, i);
            CP_ASYNC_CG(sm, reinterpret_cast<const float4*>(exp_w + row*K + col), TB);
        }
        for(int i = (threadIdx.y*blockDim.x + threadIdx.x)*TO;
                i < XS;
                i += blockDim.x*blockDim.y*TO)
        {
            int r = i/(BK*PF);
            int row = __shfl_sync(0xFFFFFFFF, token_dest[i/(XS/2)]/top_k, (r*4));
            if(row < M)
            {
                int col = b_off + i%(BK*PF);
                // if(p && block==0)
                //     printf("loading %d to tile %d %d, i %d\n", r, row, col, i);
                uint32_t sm = __cvta_generic_to_shared(s_x + i);
                CP_ASYNC_CG(sm, reinterpret_cast<const float4*>(x + row*K + col), TB);
            }
        }
        CP_ASYNC_COMMIT_GROUP();
        float scale_x[2];
        if (token_dest[0]/top_k < M)
        {
            scale_x[0] = x_scale[(token_dest[0]/top_k)*scale_cols_x + block];
        }
        if (token_dest[1]/top_k < M)
        {
            scale_x[1] = x_scale[(token_dest[1]/top_k)*scale_cols_x + block];
        }

        float scale_w = w_scale[exp_idx * scale_rows_w * scale_cols_w + (w_row/block_shape[1])*scale_cols_w + block];

        CP_ASYNC_WAIT_GROUP(0);
        __syncthreads();

        float acc[4] = {0.f};
        for(int k = 0; k < block_shape[0]; k += BK)
        {
            load_tiles(b_off + k, k/BK);
            asm volatile("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%0, %1, %2, %3};"
                    : "+f"(acc[0]), "+f"(acc[1]), "+f"(acc[2]), "+f"(acc[3])
                    : "r"(tile_x[0]), "r"(tile_x[1]), "r"(tile_x[2]), "r"(tile_x[3]), "r"(tile_w[0]), "r"(tile_w[1]));
            __syncthreads();

            // float x_dq[8];
            // float w_dq[8];
            // fp8* tmp = reinterpret_cast<fp8*>(&tile_w[0]);
            // fp8* tmp2 = reinterpret_cast<fp8*>(&tile_w[1]);
            // for (int i = 0; i < 4; i++)
            // {
            //     // x_dq[i] = float(reinterpret_cast<fp8*>(&tile_x[0])[i]) * scale_x[0];
            //     // x_dq[4 + i] = float(reinterpret_cast<fp8*>(&tile_x[2])[i]) * scale_x[0];
            //     x_dq[i] = float(reinterpret_cast<fp8*>(&tile_x[1])[i]) * scale_x[1];
            //     x_dq[4 + i] = float(reinterpret_cast<fp8*>(&tile_x[3])[i]) * scale_x[1];
            // }
            // for (int i = 0; i < 4; i++)
            // {
            //     w_dq[i] = float(tmp[i]) * scale_w;
            //     w_dq[i+4] = float(tmp2[i]) * scale_w;
            // }
            // if(p && block == 0)
            //     printf("M %d, K %d, N %d, mma %d, %d with %f,%f,%f,%f ||| %f, %f, %f, %f , w %f,%f,%f,%f ||| %f, %f, %f, %f acc %f, %f, %f, %f, scale x %f,%f scale_w %f\n",
            //             M, K, N,
            //             k,
            //             token_dest[1],
            //             x_dq[0],
            //             x_dq[1],
            //             x_dq[2],
            //             x_dq[3],
            //             x_dq[4],
            //             x_dq[5],
            //             x_dq[6],
            //             x_dq[7],
            //             w_dq[0],
            //             w_dq[1],
            //             w_dq[2],
            //             w_dq[3],
            //             float(tmp[0]),
            //             float(tmp[1]),
            //             float(tmp[2]),
            //             float(tmp[3]),
            //             acc[0],
            //             acc[1],
            //             acc[2],
            //             acc[3],
            //             scale_x[0],
            //             scale_x[1],
            //             scale_w
                        // );
        }
        if (token_dest[0]/top_k < M)
        {
            f_acc[0] += scale_x[0] * scale_w * acc[0];
            f_acc[1] += scale_x[0] * scale_w * acc[1];
        }
        if (token_dest[1]/top_k < M)
        {
            f_acc[2] += scale_x[1] * scale_w * acc[2];
            f_acc[3] += scale_x[1] * scale_w * acc[3];
        }
    }
    if (token_dest[0]/top_k < M)
    {
        *reinterpret_cast<__hip_bfloat162*>(out + token_dest[0]*N + warpN * BN + (lane_id%4)*2) = __hip_bfloat162(f_acc[0], f_acc[1]);;
    }
    if (token_dest[1]/top_k < M)
    {
        *reinterpret_cast<__hip_bfloat162*>(out + token_dest[1]*N + warpN * BN + (lane_id%4)*2) = __hip_bfloat162(f_acc[2], f_acc[3]);;
    }
    // if((token_dest[0] == 8 || token_dest[1] == 8) && (warpN*BN + (lane_id%4)*2) == 20)
    //     printf("finished with src %d/%d, dest %d/%d, off %d, exp %d, exp_off %d, %f,%f,%f,%f, b %d/%d, t %d/%d\n", token_dest[0], token_dest[1],
    //             token_dest[0], token_dest[1],
    //             warpN * BN + (lane_id%4)*2,
    //             exp_idx, exp_idx * K * N,
    //             f_acc[0],
    //             f_acc[1],
    //             f_acc[2],
    //             f_acc[3],
    //             blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

void fused_moe_w8a8_smem(
        const fp8* x,
        const float* x_scale,
        const fp8* w, const float* w_scale,
        __hip_bfloat16* out,
        const int* sorted_token_ids,
        const int* expert_ids,
        const int* num_tokens_post_padded,
        const int top_k,
        int M,
        int K,
        int N,
        int sorted_num
        )
{
    constexpr int BM = 16;
    constexpr int BK = 32;
    constexpr int BN = 8;
    constexpr int PF = 4;
    constexpr int WN = 4;
    // TODO this will only work for num_warps_y = 1
    constexpr int WM = 1;
    dim3 dimBlock(32*WN, WM, 1);
    dim3 dimGrid(std::ceil((float)N/(BN*WN)), std::ceil((float)sorted_num/(BM*WM)), 1);

    // CUtensorMap tensor_map{};
    // constexpr uint32_t rank = 3;
    // uint64_t size[rank] = {};

    fused_moe_w8a8_smem_kernel<BM, BK, BN, PF, WM, WN><<<dimGrid, dimBlock>>>(
            x,
            x_scale,
            w,
            w_scale,
            out,
            sorted_token_ids,
            expert_ids,
            num_tokens_post_padded,
            top_k,
            M,
            K,
            N
            );
}
