#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>
#include <stdio.h>

// Not gonna type all that
using fp8 = __hip_fp8_e4m3_fnuz;

#define CP_ASYNC_CG(dst, src, Bytes) \
    asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(Bytes))

#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)

#define CP_ASYNC_WAIT_GROUP(N) asm volatile("cp.async.wait_group %0;\n" ::"n"(N))

__device__ __forceinline__ void ld_matrix_x2(uint32_t* tile, uint32_t mat)
{
    asm volatile("ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0, %1}, [%2];"
            : "=r"(tile[0]), "=r"(tile[1]) : "r"(mat));
}

__device__ __forceinline__ void ld_matrix_x4(uint32_t* tile, uint32_t mat)
{
    asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0, %1, %2, %3}, [%4];"
            : "=r"(tile[0]), "=r"(tile[1]), "=r"(tile[2]), "=r"(tile[3]) : "r"(mat));
}

template <int BM, int BK, int BN, int PF, int WM, int WN>
__global__ void fused_moe_w8a8_smem_kernel(
        const fp8* __restrict__ x,
        const float* __restrict__ x_scale,
        const fp8* __restrict__ w,
        const float* __restrict__ w_scale,
        __hip_bfloat16* __restrict__ out,
        const int* __restrict__ sorted_token_ids,
        const int* __restrict__ expert_ids,
        const int* __restrict__ num_tokens_post_padded,
        const int top_k,
        int M,
        int K,
        int N
        )
{
    const int32_t warpN = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpM = blockIdx.y*blockDim.y+threadIdx.y;

    //TODO should not be hardcoded
    constexpr int block_shape[2] = {128, 128};

    const int exp_idx = expert_ids[warpM];
    const fp8* exp_w = w + exp_idx * K * N;
    const int lane_id = threadIdx.x%32;
    const int w_row = warpN * BN + (lane_id>>2);

    if(warpM * BM >= num_tokens_post_padded[0])
        return;

    // if(exp_idx < 0 || exp_idx >= 257)
    //     printf("INVALID IDX %d, %d, %d\n",blockIdx.y, exp_idx, num_tokens_post_padded[0]);


    int token_dest[2];
    token_dest[0] = sorted_token_ids[warpM*BM + (lane_id>>2)];
    token_dest[1] = sorted_token_ids[warpM*BM + (lane_id>>2) + 8];

    //SMEM sizes
    constexpr int WS = PF*BK*BN;
    constexpr int XS = PF*BK*BM;
    // how many bytes we transfer per CP_ASYNC
    constexpr int TB = 16;
    // Thread offset per transfer
    constexpr int TO = TB/sizeof(fp8);
    __shared__ alignas(128) fp8 s_w[WS];
    __shared__ alignas(128) fp8 s_x[XS];

    uint32_t tile_x[4];
    uint32_t tile_w[2];
    float f_acc[4] = {0.f};
    int compute_stage=0;
    // bool p = blockIdx.x == 0 && blockIdx.y == 64 && threadIdx.x == 0;
    auto load_tiles = [&](int off, int stage)
    {
            int xs_row = (lane_id>>2);
            if (token_dest[0]/top_k < M)
            {
                tile_x[0] = reinterpret_cast<const uint32_t*>(s_x + xs_row*PF*BK + stage*BK)[lane_id%4];
                tile_x[2] = reinterpret_cast<const uint32_t*>(s_x + xs_row*PF*BK + stage*BK + 16)[lane_id%4];
            }
            if (token_dest[1]/top_k < M)
            {
                xs_row += 8;
                tile_x[1] = reinterpret_cast<const uint32_t*>(s_x + xs_row*PF*BK + stage*BK)[lane_id%4];
                tile_x[3] = reinterpret_cast<const uint32_t*>(s_x + xs_row*PF*BK + stage*BK + 16)[lane_id%4];
            }
            // const int xs_col = (lane_id/16)*(BK/2) + stage*BK;
            // uint32_t sm_x = __cvta_generic_to_shared(s_x + xs_row*PF*BK + xs_col);
            // ld_matrix_x4(tile_x, sm_x);
            // if(p)
            //     printf("reading %d tile %d/%d, stage %d\n", token_dest[0], xs_row, xs_col, stage);

            const int ws_row = (lane_id%8);
            const int ws_col = (lane_id/8)*(BK/2) + stage*BK;
            uint32_t sm_w = __cvta_generic_to_shared(s_w + ws_row*PF*BK + ws_col);
            ld_matrix_x2(tile_w, sm_w);
    };

    for (int block=0; block < K/block_shape[0]; block += 1)
    {
        const int scale_cols_x = K/block_shape[1];
        const int scale_rows_w = N/block_shape[1];
        const int scale_cols_w = K/block_shape[0];
        int b_off = block * block_shape[0];

        for(int i = (threadIdx.y*blockDim.x + threadIdx.x)*TO;
                i < WS;
                i += blockDim.x*blockDim.y*TO)
        {
            int row = warpN*BN + i/(BK*PF);
            int col = b_off + i%(BK*PF);
            uint32_t sm = __cvta_generic_to_shared(s_w + i);
            // if(p)
            //     printf("loading %d to tile %d %d, i %d\n", exp_idx, row, col, i);
            CP_ASYNC_CG(sm, reinterpret_cast<const float4*>(exp_w + row*K + col), TB);
        }
        for(int i = (threadIdx.y*blockDim.x + threadIdx.x)*TO;
                i < XS;
                i += blockDim.x*blockDim.y*TO)
        {
            int r = i/(BK*PF);
            int row = __shfl_sync(0xFFFFFFFF, token_dest[i/(XS/2)]/top_k, r*4);
            if(row < M)
            {
                int col = b_off + i%(BK*PF);
                uint32_t sm = __cvta_generic_to_shared(s_x + i);
                CP_ASYNC_CG(sm, reinterpret_cast<const float4*>(x + row*K + col), TB);
            }
        }
        CP_ASYNC_COMMIT_GROUP();
        float scale_x[2];
        if (token_dest[0]/top_k < M)
        {
            scale_x[0] = x_scale[(token_dest[0]/top_k)*scale_cols_x + block];
        }
        if (token_dest[1]/top_k < M)
        {
            scale_x[1] = x_scale[(token_dest[1]/top_k)*scale_cols_x + block];
        }

        float scale_w = w_scale[exp_idx * scale_rows_w * scale_cols_w + (w_row/block_shape[1])*scale_cols_w + block];

        CP_ASYNC_WAIT_GROUP(0);
        __syncthreads();

        float acc[4] = {0.f};
        for(int k = 0; k < block_shape[0]; k += BK)
        {
            load_tiles(b_off + k, k/BK);
            asm volatile("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%0, %1, %2, %3};"
                    : "+f"(acc[0]), "+f"(acc[1]), "+f"(acc[2]), "+f"(acc[3])
                    : "r"(tile_x[0]), "r"(tile_x[1]), "r"(tile_x[2]), "r"(tile_x[3]), "r"(tile_w[0]), "r"(tile_w[1]));
        }
        if (token_dest[0]/top_k < M)
        {
            f_acc[0] += scale_x[0] * scale_w * acc[0];
            f_acc[1] += scale_x[0] * scale_w * acc[1];
        }
        if (token_dest[1]/top_k < M)
        {
            f_acc[2] += scale_x[1] * scale_w * acc[2];
            f_acc[3] += scale_x[1] * scale_w * acc[3];
        }
    }
    if (token_dest[0]/top_k < M)
    {
        *reinterpret_cast<__hip_bfloat162*>(out + token_dest[0]*N + warpN * BN + (lane_id%4)*2) = __hip_bfloat162(f_acc[0], f_acc[1]);;
    }
    if (token_dest[1]/top_k < M)
    {
        *reinterpret_cast<__hip_bfloat162*>(out + token_dest[1]*N + warpN * BN + (lane_id%4)*2) = __hip_bfloat162(f_acc[2], f_acc[3]);;
    }
}

void fused_moe_w8a8_smem(
        const fp8* x,
        const float* x_scale,
        const fp8* w, const float* w_scale,
        __hip_bfloat16* out,
        const int* sorted_token_ids,
        const int* expert_ids,
        const int* num_tokens_post_padded,
        const int top_k,
        int M,
        int K,
        int N,
        int sorted_num
        )
{
    constexpr int BM = 16;
    constexpr int BK = 32;
    constexpr int BN = 8;
    constexpr int PF = 4;
    constexpr int num_warps_x = 1;
    // TODO this will only work for num_warps_y = 1
    constexpr int num_warps_y = 1;
    dim3 dimBlock(32*num_warps_x, num_warps_y, 1);
    dim3 dimGrid(std::ceil((float)N/(BN*num_warps_x)), std::ceil((float)sorted_num/(BM*num_warps_y)), 1);

    // CUtensorMap tensor_map{};
    // constexpr uint32_t rank = 3;
    // uint64_t size[rank] = {};

    fused_moe_w8a8_smem_kernel<BM, BK, BN, PF, 1, 1><<<dimGrid, dimBlock>>>(
            x,
            x_scale,
            w,
            w_scale,
            out,
            sorted_token_ids,
            expert_ids,
            num_tokens_post_padded,
            top_k,
            M,
            K,
            N
            );
}
