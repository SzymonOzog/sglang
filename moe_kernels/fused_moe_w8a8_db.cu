
#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>
#include <stdio.h>

// Not gonna type all that
using fp8 = __hip_fp8_e4m3_fnuz;

#define CP_ASYNC_CG(dst, src, Bytes) \
    asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(Bytes))

#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)

#define CP_ASYNC_WAIT_GROUP(N) asm volatile("cp.async.wait_group %0;\n" ::"n"(N))

__device__ __forceinline__ void ld_matrix_x2(uint32_t* tile, uint32_t mat)
{
    asm volatile("ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0, %1}, [%2];"
            : "=r"(tile[0]), "=r"(tile[1]) : "r"(mat));
}

__device__ __forceinline__ void ld_matrix_x4(uint32_t* tile, uint32_t mat)
{
    asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0, %1, %2, %3}, [%4];"
            : "=r"(tile[0]), "=r"(tile[1]), "=r"(tile[2]), "=r"(tile[3]) : "r"(mat));
}

#define S_BITS 3
#define S_MASK 0b1110000

template <int BM, int BK, int BN, int PF, int WM, int WN, int STAGES>
__global__ void fused_moe_w8a8_db_kernel(
        const fp8* __restrict__ x,
        const float* __restrict__ x_scale,
        const fp8* __restrict__ w,
        const float* __restrict__ w_scale,
        __hip_bfloat16* __restrict__ out,
        const int* __restrict__ sorted_token_ids,
        const int* __restrict__ expert_ids,
        const int* __restrict__ num_tokens_post_padded,
        const int top_k,
        int M,
        int K,
        int N
        )
{
    const int32_t warpN = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpM = blockIdx.y*blockDim.y+threadIdx.y;

    //TODO should not be hardcoded
    constexpr int block_shape[2] = {128, 128};

    const int exp_idx = expert_ids[warpM];
    const fp8* exp_w = w + exp_idx * K * N;
    const int lane_id = threadIdx.x%32;
    const int warp_id = threadIdx.x/32;
    const int w_row = warpN * BN + (lane_id>>2);

    if(warpM * BM >= num_tokens_post_padded[0])
        return;

    // if(exp_idx < 0 || exp_idx >= 257)
    //     printf("INVALID IDX %d, %d, %d\n",blockIdx.y, exp_idx, num_tokens_post_padded[0]);


    int token_dest[2];
    token_dest[0] = sorted_token_ids[warpM*BM + (lane_id>>2)];
    token_dest[1] = sorted_token_ids[warpM*BM + (lane_id>>2) + 8];

    int token_src[2];
    token_src[0] = token_dest[0]/top_k;
    token_src[1] = token_dest[1]/top_k;

    //SMEM sizes
    constexpr int WS = WN*PF*BK*BN;
    constexpr int XS = PF*BK*BM;
    // how many bytes we transfer per CP_ASYNC
    constexpr int TB = 16;
    // Thread offset per transfer
    constexpr int TO = TB/sizeof(fp8);
    __shared__ alignas(128) fp8 s_w[STAGES*WS];
    __shared__ alignas(128) fp8 s_x[STAGES*XS];

    uint32_t tile_x[4];
    uint32_t tile_w[4];
    float f_acc[4] = {0.f};
    int compute_stage=0;
    int load_stage=0;
    int n_stages = K/block_shape[0];

    // bool p = blockIdx.x == 1 && blockIdx.y == 128 && lane_id == 14;
    // bool p = blockIdx.x == 0 && blockIdx.y == 0 && lane_id == 0;

    auto load_tiles_x = [&](int stage)
    {
        const int smem_stage = compute_stage%STAGES;
        int xs_row = (lane_id>>2);
        if (token_src[0] < M)
        {
            tile_x[0] = reinterpret_cast<const uint32_t*>(s_x + smem_stage*XS + xs_row*PF*BK + stage*BK)[lane_id%4];
            tile_x[2] = reinterpret_cast<const uint32_t*>(s_x + smem_stage*XS + xs_row*PF*BK + stage*BK + 16)[lane_id%4];
        }
        if (token_src[1] < M)
        {
            xs_row += 8;
            tile_x[1] = reinterpret_cast<const uint32_t*>(s_x + smem_stage*XS + xs_row*PF*BK + stage*BK)[lane_id%4];
            tile_x[3] = reinterpret_cast<const uint32_t*>(s_x + smem_stage*XS + xs_row*PF*BK + stage*BK + 16)[lane_id%4];
        }
    };

    auto load_tiles_w = [&](int stage)
    {
        const int ws_row = warp_id*BN + (lane_id%8);
        const int ws_col = (lane_id/8)*(BK/2) + stage*BK;
        const int smem_stage = compute_stage%STAGES;

        int i = ws_row*PF*BK + ws_col;
        int swizzled = i^((i&(S_MASK<<S_BITS))>>S_BITS);
        uint32_t sm_w = __cvta_generic_to_shared(s_w + smem_stage*WS + swizzled);
        ld_matrix_x4(tile_w, sm_w);
    };


    auto async_load = [&]()
    {
        const int off = load_stage * block_shape[0];
        int smem_stage = load_stage%STAGES;
        for(int i = (threadIdx.y*blockDim.x + threadIdx.x)*TO;
                i < WS;
                i += blockDim.x*blockDim.y*TO)
        {
            int row = blockIdx.x*WN*BN + i/(BK*PF);
            int col = off + i%(BK*PF);
            int swizzled = i^((i&(S_MASK<<S_BITS))>>S_BITS);
            uint32_t sm = __cvta_generic_to_shared(s_w + smem_stage*WS + swizzled);
            CP_ASYNC_CG(sm, reinterpret_cast<const float4*>(exp_w + row*K + col), TB);
        }

        for(int i = (threadIdx.y*blockDim.x + threadIdx.x)*TO;
                i < XS;
                i += blockDim.x*blockDim.y*TO)
        {
            int r = i/(BK*PF);
            int tok_dst = i/(XS/2) == 0 ? token_dest[0] : token_dest[1];
            int row = __shfl_sync(0xFFFFFFFF, tok_dst/top_k, (r*4));
            if(row < M)
            {
                int col = off + i%(BK*PF);
                uint32_t sm = __cvta_generic_to_shared(s_x + smem_stage*XS + i);
                CP_ASYNC_CG(sm, reinterpret_cast<const float4*>(x + row*K + col), TB);
            }
        }
        CP_ASYNC_COMMIT_GROUP();
        load_stage++;
    };

    async_load();

    for (int block=0; block < n_stages; block += 1)
    {
        compute_stage = block;
        const int scale_cols_x = K/block_shape[1];
        const int scale_rows_w = N/block_shape[1];
        const int scale_cols_w = K/block_shape[0];

        float scale_x[2];
        if (token_src[0] < M)
        {
            scale_x[0] = x_scale[(token_src[0])*scale_cols_x + block];
        }
        if (token_src[1] < M)
        {
            scale_x[1] = x_scale[(token_src[1])*scale_cols_x + block];
        }

        float scale_w = w_scale[exp_idx * scale_rows_w * scale_cols_w + (w_row/block_shape[1])*scale_cols_w + block];

        CP_ASYNC_WAIT_GROUP(0);
        __syncthreads();

        if(load_stage < n_stages)
        {
            async_load();
        }

        float acc[4] = {0.f};
        load_tiles_x(0);
        load_tiles_w(0);
        asm volatile("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%0, %1, %2, %3};"
                : "+f"(acc[0]), "+f"(acc[1]), "+f"(acc[2]), "+f"(acc[3])
                : "r"(tile_x[0]), "r"(tile_x[1]), "r"(tile_x[2]), "r"(tile_x[3]), "r"(tile_w[0]), "r"(tile_w[1]));
        load_tiles_x(1);
        asm volatile("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%0, %1, %2, %3};"
                : "+f"(acc[0]), "+f"(acc[1]), "+f"(acc[2]), "+f"(acc[3])
                : "r"(tile_x[0]), "r"(tile_x[1]), "r"(tile_x[2]), "r"(tile_x[3]), "r"(tile_w[2]), "r"(tile_w[3]));
        load_tiles_x(2);
        load_tiles_w(2);
        asm volatile("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%0, %1, %2, %3};"
                : "+f"(acc[0]), "+f"(acc[1]), "+f"(acc[2]), "+f"(acc[3])
                : "r"(tile_x[0]), "r"(tile_x[1]), "r"(tile_x[2]), "r"(tile_x[3]), "r"(tile_w[0]), "r"(tile_w[1]));
        load_tiles_x(3);
        asm volatile("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%0, %1, %2, %3};"
                : "+f"(acc[0]), "+f"(acc[1]), "+f"(acc[2]), "+f"(acc[3])
                : "r"(tile_x[0]), "r"(tile_x[1]), "r"(tile_x[2]), "r"(tile_x[3]), "r"(tile_w[2]), "r"(tile_w[3]));

        if (token_src[0] < M)
        {
            f_acc[0] += scale_x[0] * scale_w * acc[0];
            f_acc[1] += scale_x[0] * scale_w * acc[1];
        }
        if (token_src[1] < M)
        {
            f_acc[2] += scale_x[1] * scale_w * acc[2];
            f_acc[3] += scale_x[1] * scale_w * acc[3];
        }
    }
    if (token_src[0] < M)
    {
        *reinterpret_cast<__hip_bfloat162*>(out + token_dest[0]*N + warpN * BN + (lane_id%4)*2) = __hip_bfloat162(f_acc[0], f_acc[1]);;
    }
    if (token_src[1] < M)
    {
        *reinterpret_cast<__hip_bfloat162*>(out + token_dest[1]*N + warpN * BN + (lane_id%4)*2) = __hip_bfloat162(f_acc[2], f_acc[3]);;
    }

    // if((token_dest[0] == 8 || token_dest[1] == 8) && (warpN*BN + (lane_id%4)*2) == 20)
    //     printf("finished with src %d/%d, dest %d/%d, off %d, exp %d, exp_off %d, %f,%f,%f,%f, b %d/%d, t %d/%d\n", token_dest[0], token_dest[1],
    //             token_dest[0], token_dest[1],
    //             warpN * BN + (lane_id%4)*2,
    //             exp_idx, exp_idx * K * N,
    //             f_acc[0],
    //             f_acc[1],
    //             f_acc[2],
    //             f_acc[3],
    //             blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);

}

void fused_moe_w8a8_db(
        const fp8* x,
        const float* x_scale,
        const fp8* w, const float* w_scale,
        __hip_bfloat16* out,
        const int* sorted_token_ids,
        const int* expert_ids,
        const int* num_tokens_post_padded,
        const int top_k,
        int M,
        int K,
        int N,
        int sorted_num
        )
{
    constexpr int BM = 16;
    constexpr int BK = 32;
    constexpr int BN = 8;
    constexpr int PF = 4;
    constexpr int WN = 8;
    // TODO this will only work for num_warps_y = 1
    constexpr int WM = 1;
    dim3 dimBlock(32*WN, WM, 1);
    dim3 dimGrid(std::ceil((float)N/(BN*WN)), std::ceil((float)sorted_num/(BM*WM)), 1);

    fused_moe_w8a8_db_kernel<BM, BK, BN, PF, WM, WN, 2><<<dimGrid, dimBlock>>>(
            x,
            x_scale,
            w,
            w_scale,
            out,
            sorted_token_ids,
            expert_ids,
            num_tokens_post_padded,
            top_k,
            M,
            K,
            N
            );
}
            // float x_dq[8];
            // float w_dq[8];
            // fp8* tmp = reinterpret_cast<fp8*>(&tile_w[0]);
            // fp8* tmp2 = reinterpret_cast<fp8*>(&tile_w[1]);
            // for (int i = 0; i < 4; i++)
            // {
            //     // x_dq[i] = float(reinterpret_cast<fp8*>(&tile_x[0])[i]) * scale_x[0];
            //     // x_dq[4 + i] = float(reinterpret_cast<fp8*>(&tile_x[2])[i]) * scale_x[0];
            //     x_dq[i] = float(reinterpret_cast<fp8*>(&tile_x[1])[i]) * scale_x[1];
            //     x_dq[4 + i] = float(reinterpret_cast<fp8*>(&tile_x[3])[i]) * scale_x[1];
            // }
            // for (int i = 0; i < 4; i++)
            // {
            //     w_dq[i] = float(tmp[i]) * scale_w;
            //     w_dq[i+4] = float(tmp2[i]) * scale_w;
            // }
            // if(p && block == 0)
            //     printf("M %d, K %d, N %d cs %d, ls %d, mma %d, %d with %f,%f,%f,%f ||| %f, %f, %f, %f , w %f,%f,%f,%f ||| %f, %f, %f, %f acc %f, %f, %f, %f, scale x %f,%f scale_w %f\n",
            //             M, K, N,
            //             compute_stage, load_stage,
            //             k,
            //             token_dest[1],
            //             x_dq[0],
            //             x_dq[1],
            //             x_dq[2],
            //             x_dq[3],
            //             x_dq[4],
            //             x_dq[5],
            //             x_dq[6],
            //             x_dq[7],
            //             w_dq[0],
            //             w_dq[1],
            //             w_dq[2],
            //             w_dq[3],
            //             float(tmp[0]),
            //             float(tmp[1]),
            //             float(tmp[2]),
            //             float(tmp[3]),
            //             acc[0],
            //             acc[1],
            //             acc[2],
            //             acc[3],
            //             scale_x[0],
            //             scale_x[1],
            //             scale_w
            //             );
